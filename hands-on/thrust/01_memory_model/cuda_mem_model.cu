// C++ standard headers
#include <cassert>
#include <iostream>
#include <ranges>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

// CUDA headers
#include <hip/hip_runtime.h>

// local headers
#include "cuda_check.h"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main() {
  // Choose one CUDA device
  CUDA_CHECK(hipSetDevice(MYDEVICE));

  // Pointer and dimension for host memory
  int dim = 1024;
  // Part 1: allocate a buffer in host memory with increasing values from 0 to
  // dimA - 1
  auto h_buffer = ...;

  // Part 2: allocate two buffers in device memory
  auto d_buffer1 = ...;
  auto d_buffer2 = ...;

  // Part 3: copy the content of the host buffer to the first device buffer

  // Part 4: copy the content of the first device buffer to the second device
  // buffer

  // Part 5: set all the values in the host buffer to zero

  // Part 6: copy the contant back to the host buffer

  // Verify the data on the host is correct
  assert(std::ranges::equal(h_buffer, std::views::iota(0, dim)));

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct!" << std::endl;

  return 0;
}
